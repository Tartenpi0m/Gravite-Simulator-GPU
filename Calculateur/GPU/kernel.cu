#include "hip/hip_runtime.h"
#include <stdio.h>
#include "planete.h"
#include "simu.cuh"
#include "to_json.h"
#include "kernel.cuh"
#include "book.h"

__global__ void kernel_calculCollision(int indice, planete une_planete, planete * all_planete, int N_corps) {


    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i< N_corps && indice < i) {

        if(detect_collision(&une_planete, &all_planete[i])) {
            //printf("%d,%d\n",i,j);
            regroupe(&all_planete[i], &all_planete[indice]);
        }
    }

    /* int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.x * blockDim.x + threadIdx.y;
    
    if (i<N_corps && j<N_corps && i < j ){
        if(all_planete[i].id != -1) {
            if(all_planete[j].id != -1) {
                if(detect_collision(&all_planete[i], &all_planete[j])) {
                    printf("%d,%d\n",i,j);
                    regroupe(&all_planete[i], &all_planete[j]);
                }
            }
        }
    } */
}

void calculCollision(planete * all_planete, int N_corps) {

    for(int i = 0; i < N_corps; i++) {
        for(int j = 0; j < N_corps; j++) {
            if(i != j) {
                if(all_planete[i].id != -1) {
                    if(all_planete[j].id != -1) {
                        if(detect_collision(&all_planete[i], &all_planete[j])) {
                            regroupe(&all_planete[i], &all_planete[j]);
                        }
                    }
                } else {
                    break;
                }
            }    
        }
    }
}

__global__ void kernel_calculAcceleration(int indice, planete * all_planete, int N_corps, long int G) {

    int i = threadIdx.x + blockIdx.x*blockDim.x;

    if(i < N_corps) {
        if(i != indice && all_planete[i].id != -1 && all_planete[indice].id != -1) {  
            all_planete[i].a[0] += force_G(G, &all_planete[i], &all_planete[indice], 0);
            all_planete[i].a[1] += force_G(G, &all_planete[i], &all_planete[indice], 1); 
        }
    }

}

void calculAcceleration(planete * all_planete, int N_corps, long int G) {

    for(int i = 0; i < N_corps; i++) {
            all_planete[i].a[0] = 0;
            all_planete[i].a[1] = 0;
            for(int j = 0; j < N_corps; j++) {
                if(i!=j) {
                    if(all_planete[i].id != -1) {
                        if(all_planete[j].id != -1) {
                            all_planete[i].a[0] += force_G(G, &all_planete[i], &all_planete[j],0);
                            all_planete[i].a[1] += force_G(G, &all_planete[i], &all_planete[j],1);
                        }
                    } else {
                        break;
                    }
                }
            }
        }
}

__global__ void kernel_updatePosition(planete * all_planete, int N_corps) {

    int i = threadIdx.x + blockIdx.x*blockDim.x;
     if(i < N_corps) {
        if(all_planete[i].id >=1) {

            //Ajoute les accélérations aux vitesses
            all_planete[i].v[0] += 0.00000000001*all_planete[i].a[0]; //0.000000000001
            all_planete[i].v[1] += 0.00000000001*all_planete[i].a[1];

            //Ajoute les vitesses aux positions
            all_planete[i].x[0] += all_planete[i].v[0];
            all_planete[i].x[1] += all_planete[i].v[1];

            all_planete[i].a[0] = 0;
            all_planete[i].a[1] = 0;

            
        }
    } 
}

void gestion(int nb_frame, planete * all_planete, int N_corps, FILE * data, long int G) {

    int frame = 0;

    planete * all_planete_cuda;
    HANDLE_ERROR(hipMalloc((void **) &all_planete_cuda, sizeof(planete)*N_corps));


    int n_blocks = (int) N_corps / 1024 + 1;
    int n_blocks2 = (int) N_corps / 200 + 1;

    printf("\n\n\n%d, %d\n\n\n", n_blocks, n_blocks2);
    
    HANDLE_ERROR(hipMemcpy(all_planete_cuda, all_planete, sizeof(planete)*N_corps, hipMemcpyHostToDevice));
    while(frame < nb_frame) {

        printf("frame : %d\n", frame);
        
        for(int i = 0; i < N_corps; i++) {
            kernel_calculCollision<<<n_blocks,1024>>>(i, all_planete[i], all_planete_cuda, N_corps);      
        } 

        for(int indice = 0; indice < N_corps; indice++) {
            kernel_calculAcceleration<<<n_blocks2,200>>>(indice, all_planete_cuda, N_corps, G);
        }

        kernel_updatePosition<<<n_blocks,1024>>>(all_planete_cuda, N_corps);      


        hipDeviceSynchronize();
        HANDLE_ERROR(hipMemcpy(all_planete, all_planete_cuda, sizeof(planete)*N_corps, hipMemcpyDeviceToHost)); 
        
        write_frame(data,frame, all_planete, N_corps);
        if(frame < nb_frame-1) {
            virgule(data);
        }
        frame++;
    }

    hipFree((void*) all_planete_cuda);


}