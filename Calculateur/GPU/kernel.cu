#include "hip/hip_runtime.h"
#include <stdio.h>
#include "planete.h"
#include "simu.cuh"
#include "to_json.h"
#include "kernel.cuh"
#include "book.h"
#include <sys/time.h>

__global__ void kernel_calculCollision(int indice, planete une_planete, planete * all_planete, int N_corps) {


    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i< N_corps && indice > i) {

        if(detect_collision(&une_planete, &all_planete[i])) {
            //printf("%d,%d\n",i,j);
            regroupe(&all_planete[i], &all_planete[indice]);
        }
    }

    /* int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.x * blockDim.x + threadIdx.y;
    
    if (i<N_corps && j<N_corps && i < j ){
        if(all_planete[i].id != -1) {
            if(all_planete[j].id != -1) {
                if(detect_collision(&all_planete[i], &all_planete[j])) {
                    printf("%d,%d\n",i,j);
                    regroupe(&all_planete[i], &all_planete[j]);
                }
            }
        }
    } */
}


__global__ void kernel_calculAcceleration(int indice, planete * all_planete, int N_corps, long int G) {

    int i = threadIdx.x + blockIdx.x*blockDim.x;

    if(i < N_corps) {
        if(i != indice && all_planete[i].id != -1 && all_planete[indice].id != -1) {  
            double x = force_G(G, &all_planete[i], &all_planete[indice], 0);
            double y = force_G(G, &all_planete[i], &all_planete[indice], 1);

            if(isnan(all_planete[i].a[0]) || isnan(all_planete[i].a[1])) {
                all_planete[i].a[0] = 0.0;
                all_planete[i].a[1] = 0.0;
            }   
            
            all_planete[i].a[0] += force_G(G, &all_planete[i], &all_planete[indice], 0);
            all_planete[i].a[1] += force_G(G, &all_planete[i], &all_planete[indice], 1); 
        }
    }

}


__global__ void kernel_updatePosition(planete * all_planete, int N_corps) {

    int i = threadIdx.x + blockIdx.x*blockDim.x;
     if(i < N_corps) {
        if(all_planete[i].id >=1) {

            //Ajoute les accélérations aux vitesses
            all_planete[i].v[0] += 0.00000000001*all_planete[i].a[0]; //0.000000000001
            all_planete[i].v[1] += 0.00000000001*all_planete[i].a[1];

            //Ajoute les vitesses aux positions
            all_planete[i].x[0] += all_planete[i].v[0];
            all_planete[i].x[1] += all_planete[i].v[1];

            all_planete[i].a[0] = 0;
            all_planete[i].a[1] = 0;

            
        }
    } 
}

__global__ void kkk(planete * all_planete, int size) {

    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if(i < size) {
        if(isnan(all_planete[i].x[0])) {
            printf("\nNaN\n");
            all_planete[i].id = -1;
        }
    }
}


__host__ void gestion(int nb_frame, planete * all_planete, int N_corps, FILE * data, long int G) {

    int frame = 0;

    planete * all_planete_cuda;
    HANDLE_ERROR(hipMalloc((void **) &all_planete_cuda, sizeof(planete)*N_corps));


    int n_blocks = (int) N_corps / 1024 + 1;
    int n_blocks2 = (int) N_corps / 800 + 1;

    printf("\n\n\n%d, %d\n\n\n", n_blocks, n_blocks2);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float temps_frame;
    float temps_totale = 0;
    
    struct timeval start2;
    struct timeval end;
    float temps_Memcpy = 0;


    HANDLE_ERROR(hipMemcpy(all_planete_cuda, all_planete, sizeof(planete)*N_corps, hipMemcpyHostToDevice));
    while(frame < nb_frame) {

        printf("frame : %d\n", frame);


        ////////////////KERNEL////////////////////////
        hipEventRecord(start);
        for(int i = 0; i < N_corps; i++) {
            kernel_calculCollision<<<n_blocks,1024>>>(i, all_planete[i], all_planete_cuda, N_corps);      
        } 

        for(int indice = 0; indice < N_corps; indice++) {
            kernel_calculAcceleration<<<n_blocks2,800>>>(indice, all_planete_cuda, N_corps, G);
        }

        kernel_updatePosition<<<n_blocks,1024>>>(all_planete_cuda, N_corps);      
        hipEventRecord(stop);

        //kkk<<<n_blocks, 1024>>>(all_planete_cuda, N_corps);
        hipDeviceSynchronize();
        hipEventElapsedTime(&temps_frame, start, stop);
        temps_totale += temps_frame;



        ////////////MEMCPY//////////////////////
        gettimeofday(&start2, NULL);
        HANDLE_ERROR(hipMemcpy(all_planete, all_planete_cuda, sizeof(planete)*N_corps, hipMemcpyDeviceToHost)); 
        gettimeofday(&end, NULL);
        temps_Memcpy += (end.tv_sec - start2.tv_sec) + 1e-6*(end.tv_usec - start2.tv_usec);
    

        ///////////JSON/////////////////
        write_frame(data,frame, all_planete, N_corps);
        if(frame < nb_frame-1) {
            virgule(data);
        }


        frame++;
    }



    hipFree((void*) all_planete_cuda);



    printf("\n\n\n KERNEL TIME : %lf s\n",  temps_totale/1000);
    printf("\n MEMCPY TIME : %f s\n\n\n", temps_Memcpy);


}