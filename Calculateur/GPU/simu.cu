#include "hip/hip_runtime.h"
#include "planete.h"
#include "simu.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__device__ double mini(double a, double b) {
    if(a < b) {
        return a;
    } else {
        return b;
    }
}

__host__ __device__ double distance(planete * a, planete * b) {
    return sqrt(pow(fabs(a->x[0] - b->x[0]),2) + pow(fabs(a->x[1] - b->x[1]),2));
}

//Regroupe 2 planetes en une (moyenne des position, moyenne des vitesses, addition des volumes (aires))
__host__ __device__ void regroupe(planete * a, planete* b) {


    if(a->id == -2) { //Si je trou noir, je retrecis l'autre
        b->id = -1;
        b->rayon = 0;
        b->v[0] = 0;
        b->v[1] = 0;
        b->x[0] = 0;
        b->x[1] = 0;
        b->masse = 0;

    } else if (b->id == -2) {
        a->id = -1;
        a->rayon = 0;
        a->v[0] = 0;
        a->v[1] = 0;
        a->x[0] = 0;
        a->x[1] = 0;
        a->masse = 0;

    } else { //Si tt le monde est planete

        if(a->rayon < b->rayon) {
            planete * tmp = a;
            a = b;
            b = tmp;
        } 

        a->v[0] = (a->v[0] +b->v[0]) / 2;
        a->v[1] = (a->v[1] + b->v[1]) / 2; 

        double ratio = a->masse/b->masse;
        double coefa = 2*(ratio/(ratio+1));
        a->x[0] = (coefa*a->x[0] + (2-coefa)*b->x[0])/2;
        a->x[1] = (coefa*a->x[1] + (2-coefa)*b->x[1])/2;
        a->rayon = sqrt(pow(a->rayon,2)+pow(b->rayon,2));
        a->masse += b->masse;

        b->id = -1;
        b->rayon = 0;
        b->v[0] = 0;
        b->v[1] = 0;
        b->x[0] = 0;
        b->x[1] = 0;
        b->masse = 0;
    
    }


}

__host__ __device__ short detect_collision(planete * a, planete * b) {

    //Si collison
    if( distance(a,b) < a->rayon + b->rayon) {
        return 1; //return True
    } else {
        return 0;
    }
}

__host__ __device__ double force_G(long int G, planete * a, planete * b, int coord) {
    double d_h = (a->x[0] - b->x[0]);
    double d_v = (a->x[1] - b->x[1]);
    double d = distance(a, b);
    double force = G*pow(10,-11)*(b->masse/(d*d));
    double teta = acos(fabs(d_h/d));

    double x;
    if(coord == 0) {
        x = cos(teta)*force;
        if(d_h < 0) {
            return x;
        } else {
            return -x;
        }
    } else {
        x = sin(teta)*force; 
        if(d_v > 0) {
            return -x;
        }  else {
            return x;
        }
    }
    
    

    

    
}