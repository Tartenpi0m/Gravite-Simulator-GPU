#include "hip/hip_runtime.h"
#include <stdio.h>
#include "planete.h"
#include "simu.cuh"
#include "to_json.h"
#include "book.h"

__global__ void kernel_calculCollision(planete ** all_planete, int N_corps) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.x * blockDim.x + threadIdx.y;
    
    if (i<N_corps && j<N_corps){
        printf("%d,%d\n",i,j);
        if(all_planete[i]->id != -1) {
            if(all_planete[j]->id != -1) {
                if(detect_collision(all_planete[i], all_planete[j])) {
                    regroupe(all_planete[i], all_planete[j]);
                }
            }
        }
    }
}

void calculCollision(planete ** all_planete, int N_corps) {

    for(int i = 0; i < N_corps; i++) {
        for(int j = 0; j < N_corps; j++) {
            if(i != j) {
                if(all_planete[i]->id != -1) {
                    if(all_planete[j]->id != -1) {
                        if(detect_collision(all_planete[i], all_planete[j])) {
                            regroupe(all_planete[i], all_planete[j]);
                        }
                    }
                } else {
                    break;
                }
            }    
        }
    }
}


void calculAcceleration(planete ** all_planete, int N_corps, long int G) {

    for(int i = 0; i < N_corps; i++) {
            all_planete[i]->a[0] = 0;
            all_planete[i]->a[1] = 0;
            for(int j = 0; j < N_corps; j++) {
                if(i!=j) {
                    if(all_planete[i]->id != -1) {
                        if(all_planete[j]->id != -1) {
                            all_planete[i]->a[0] += force_G(G, all_planete[i], all_planete[j],0);
                            all_planete[i]->a[1] += force_G(G, all_planete[i], all_planete[j],1);
                        }
                    } else {
                        break;
                    }
                }
            }
        }
}

__global__ void kernel_updatePosition(planete ** all_planete, int N_corps) {

    int i = threadIdx.x + blockIdx.x*blockDim.x;
    all_planete[0]->id = 5;
    /* if(i < N_corps) {
        if(all_planete[i]->id >=1) {

            //Ajoute les accélérations aux vitesses
            all_planete[i]->v[0] += 0.00000000001*all_planete[i]->a[0]; //0.000000000001
            all_planete[i]->v[1] += 0.00000000001*all_planete[i]->a[1];

            //Ajoute les vitesses aux positions
            all_planete[i]->x[0] += all_planete[i]->v[0];
            all_planete[i]->x[1] += all_planete[i]->v[1];
        }
    } */
}


void gestion(int nb_frame, planete ** all_planete, int N_corps, FILE * data, long int G) {

    int frame = 0;
    
    /* int * N_corps_cuda;
    hipMalloc((void **) &N_corps_cuda, sizeof(int));
    hipMemcpy(N_corps_cuda, &N_corps, sizeof(int), hipMemcpyHostToDevice); */

    planete ** all_planete_cuda;
    HANDLE_ERROR(hipMalloc((void ***) &all_planete_cuda, sizeof(planete*)*N_corps));
    for(int i = 0; i < N_corps; i++) {
        HANDLE_ERROR(hipMalloc((void **) &all_planete_cuda[i], sizeof(planete)));
    } 


    int n_blocks_calculCollision= (int) (N_corps*N_corps) / 1024 + 1;
    dim3 thread_per_blocks(32,32);

    int n_blocks_update_position = (int) N_corps / 1024 +1;
    printf("\n//%d // %d\n", n_blocks_calculCollision, n_blocks_update_position);
    //planete tab_planete[N_corps]; 
    
    while(frame < nb_frame) {
        //printf("Check collison");

        //HANDLE_ERROR(hipMemcpy(all_planete_cuda, all_planete, sizeof(planete*)*N_corps, hipMemcpyHostToDevice));
        //kernel_calculCollision<<<n_blocks_calculCollision,thread_per_blocks>>>(all_planete_cuda, N_corps);
        //HANDLE_ERROR(hipMemcpy(all_planete, all_planete_cuda, sizeof(planete*)*N_corps, hipMemcpyDeviceToHost));
        //hipDeviceSynchronize();
        calculCollision(all_planete, N_corps);
        calculAcceleration(all_planete, N_corps, G);

        HANDLE_ERROR(hipMemcpy(all_planete_cuda, all_planete, sizeof(planete*)*N_corps, hipMemcpyHostToDevice));
        for(int i = 0; i < N_corps; i++) {
            HANDLE_ERROR(hipMemcpy(all_planete_cuda[i], all_planete[i], sizeof(planete), hipMemcpyHostToDevice));
        } 
        
        kernel_updatePosition<<<n_blocks_update_position,1024>>>(all_planete_cuda, N_corps);
        
        HANDLE_ERROR(hipMemcpy(all_planete, all_planete_cuda, sizeof(planete*)*N_corps, hipMemcpyDeviceToHost)); 
        for(int i = 0; i < N_corps; i++) {
            HANDLE_ERROR(hipMemcpy(all_planete[i], all_planete_cuda[i], sizeof(planete), hipMemcpyDeviceToHost));
        } 
        


        printf(" x2:%lf\n", all_planete[0]->x[0]);

        write_frame(data,frame, all_planete, N_corps);
        if(frame < nb_frame-1) {
            virgule(data);
        }

        frame++;
    }

    hipFree((void*) all_planete_cuda);

}